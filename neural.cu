/**
 * @file neural_cuda.cpp
 * @author Michael Gathara (michael@michaelgathara.com)
 * @brief 
 * @version 0.1
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include <iostream>
#include <vector>
#include <cmath>
#include <random>
#include <hip/hip_runtime.h>

__device__ 
double sigmoid(double x) {
    return 1.0 / (1.0 + exp(-x));
}

__global__ 
void hiddenLayerKernel(double* g_input, double* g_hidden_weights, double* g_hidden_values, double hidden_bias, int input_size, int hidden_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < hidden_size) {
        double value = 0.0;
        for (int j = 0; j < input_size; j++) {
            value += g_input[j] * g_hidden_weights[i * input_size + j];
        }
        value += hidden_bias;
        g_hidden_values[i] = sigmoid(value);
    }
}

__global__ 
void outputLayerKernel(double* g_hidden_values, double* g_output_weights, double* g_output_value, double output_bias, int hidden_size) {
    double value = 0.0;
    for (int i = 0; i < hidden_size; i++) {
        value += g_hidden_values[i] * g_output_weights[i];
    }
    value += output_bias;
    *g_output_value = sigmoid(value);
}

class NeuralNetwork {
private:
    std::vector<std::vector<double>> hidden_weights;
    std::vector<double> output_weights;
    double hidden_bias;
    double output_bias;
    std::mt19937 gen;
    std::uniform_real_distribution<> dis;
    double* g_hidden_weights;
    double* g_output_weights;

public:
    NeuralNetwork(int input_nodes, int hidden_nodes, int output_nodes) : gen(std::random_device{}()), dis(-1.0, 1.0) {
        hidden_weights.resize(hidden_nodes, std::vector<double>(input_nodes));
        for (int i = 0; i < hidden_nodes; i++) {
            for (int j = 0; j < input_nodes; j++) {
                hidden_weights[i][j] = dis(gen);
            }
        }

        output_weights.resize(output_nodes);
        for (int i = 0; i < output_nodes; i++) {
            output_weights[i] = dis(gen);
        }

        hidden_bias = dis(gen);
        output_bias = dis(gen);

        hipMalloc(&g_hidden_weights, hidden_nodes * input_nodes * sizeof(double));
        hipMalloc(&g_output_weights, output_nodes * sizeof(double));

        hipMemcpy(g_hidden_weights, hidden_weights.data(), hidden_nodes * input_nodes * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(g_output_weights, output_weights.data(), output_nodes * sizeof(double), hipMemcpyHostToDevice);
    }


    std::vector<double> forward(const std::vector<double>& input) {
        double* g_input;
        double* g_hidden_values;
        double* g_output_value;

        hipMalloc(&g_input, input.size() * sizeof(double));
        hipMalloc(&g_hidden_values, hidden_weights.size() * sizeof(double));
        hipMalloc(&g_output_value, sizeof(double));

        hipMemcpy(g_input, input.data(), input.size() * sizeof(double), hipMemcpyHostToDevice);

        hiddenLayerKernel<<<(hidden_weights.size() + 255) / 256, 256>>>(g_input, g_hidden_weights, g_hidden_values, hidden_bias, input.size(), hidden_weights.size());
        hipDeviceSynchronize();

        outputLayerKernel<<<1, 1>>>(g_hidden_values, g_output_weights, g_output_value, output_bias, hidden_weights.size());
        hipDeviceSynchronize();

        double output_value;
        hipMemcpy(&output_value, g_output_value, sizeof(double), hipMemcpyDeviceToHost);

        hipFree(g_input);
        hipFree(g_hidden_values);
        hipFree(g_output_value);

        return {output_value};
    }

    ~NeuralNetwork() {
        hipFree(g_hidden_weights);
        hipFree(g_output_weights);
    }
};

int main() {
    NeuralNetwork nn(5, 10, 1);
    std::vector<double> input = {0.5, 0.3, 0.2, 0.2, 0.1};
    std::vector<double> output = nn.forward(input);

    std::cout << "Output: " << output[0] << std::endl;

    return 0;
}
